#include "hip/hip_runtime.h"
#include <fft/fft.hpp>

#define BLOCK_SIZE 32

using real = double;

__device__ static void radix2(double* X, double* Y, int N, int M) {
	const int& tid = threadIdx.x;
	const int N2 = N >> 1;
	const int MN2 = M * N2;
	if (N > 2) {
		radix2(X, Y, N2, M);
		radix2(X + MN2, Y + MN2, N2, M);
	}
	const real twopioN = real(-2.0 * M_PI) / N;
	for (int k2 = 0; k2 < N2; k2++) {
		const real phi = twopioN * k2;
		const real cos1 = cos(phi);
		const real sin1 = sin(phi);
		for (int m = tid; m < M; m += BLOCK_SIZE) {
			const int& i0 = M * k2 + m;
			const int i1 = i0 + MN2;
			real& er0 = X[i0];
			real& er1 = X[i1];
			real& ei0 = Y[i0];
			real& ei1 = Y[i1];
			real tr1 = er1;
			er1 = er0 - cos1 * tr1 + ei1 * sin1;
			ei1 = ei0 - sin1 * tr1 - ei1 * cos1;
			er0 += er0 - er1;
			ei0 += ei0 - ei1;
		}
	}
}

__global__ void fft_1d_kernel(double* X, double* Y, int N, int M) {
	radix2(X, Y, N, M);
}

void cuda_fft_1d(double* Xh, double* Yh, int N, int M) {
	real* Xd;
	real* Yd;
	const size_t size = N * M * sizeof(real);
	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));
	CUDA_CHECK(hipMallocAsync(&Xd, size, stream));
	CUDA_CHECK(hipMallocAsync(&Yd, size, stream));
	CUDA_CHECK(hipMemcpyAsync(Xd, Xh, size, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(Yd, Yh, size, hipMemcpyHostToDevice, stream));
	fft_1d_kernel<<<1, BLOCK_SIZE, 0, stream>>>(Xd, Yd, N, M);
	CUDA_CHECK(hipMemcpyAsync(Xh, Xd, size, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(Yh, Yd, size, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipFreeAsync(Xd, stream));
	CUDA_CHECK(hipFreeAsync(Yd, stream));
	while (hipStreamQuery(stream) != hipSuccess) {
		yield();
	}
	CUDA_CHECK(hipStreamDestroy(stream));
}

